#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t err, const char* msg) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 1024;
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    float alpha = 1.0f, beta = 0.0f;

    h_A = new float[N * N];
    h_B = new float[N * N];
    h_C = new float[N * N];

    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
        h_C[i] = 0.0f;
    }

    checkCudaError(hipMalloc((void**)&d_A, N * N * sizeof(float)), "CUDA malloc A");
    checkCudaError(hipMalloc((void**)&d_B, N * N * sizeof(float)), "CUDA malloc B");
    checkCudaError(hipMalloc((void**)&d_C, N * N * sizeof(float)), "CUDA malloc C");

    checkCudaError(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice), "CUDA memcpy A");
    checkCudaError(hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice), "CUDA memcpy B");

    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "CUBLAS initialization");

    // Warm up
    checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N), "CUBLAS SGEMM");

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Event create start");
    checkCudaError(hipEventCreate(&stop), "Event create stop");

    checkCudaError(hipEventRecord(start, 0), "Event record start");
    for(int i = 0; i < 100; i++)
        checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N), "CUBLAS SGEMM");
    checkCudaError(hipEventRecord(stop, 0), "Event record stop");

    checkCudaError(hipEventSynchronize(stop), "Event synchronize stop");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Event elapsed time");

    std::cout << "cuBLAS GEMM time: " << milliseconds << " ms" << std::endl;

    // Clean up
    checkCublasError(hipblasDestroy(handle), "CUBLAS destroy");
    checkCudaError(hipFree(d_A), "CUDA free A");
    checkCudaError(hipFree(d_B), "CUDA free B");
    checkCudaError(hipFree(d_C), "CUDA free C");
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
