#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t err, const char* msg) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 3; // Size of the matrix (N x N)

    // Host memory
    float h_A[N*N] = {4, 7, 2, 3, 6, 1, 2, 5, 1}; // Input matrix
    float h_Ainv[N*N]; // Output matrix (inverse)

    // Device memory
    float* d_A;
    checkCudaError(hipMalloc((void**)&d_A, N * N * sizeof(float)), "Failed to allocate device memory for A");
    checkCudaError(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy A to device");

    int* d_P; // Pivot indices
    checkCudaError(hipMalloc((void**)&d_P, N * sizeof(int)), "Failed to allocate device memory for pivot indices");

    int* d_info; // Info output
    checkCudaError(hipMalloc((void**)&d_info, sizeof(int)), "Failed to allocate device memory for info");

    // cuBLAS handle
    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "Failed to create cuBLAS handle");

    // Perform LU decomposition
    checkCublasError(hipblasSgetrfBatched(handle, N, &d_A, N, d_P, d_info, 1), "Failed to perform LU decomposition");

    // Check for successful LU decomposition
    int h_info;
    checkCudaError(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost), "Failed to copy info to host");
    if (h_info != 0) {
        std::cerr << "LU decomposition failed: Matrix is singular" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Perform matrix inversion
    checkCublasError(hipblasSgetriBatched(handle, N, (const float**)&d_A, N, d_P, &d_A, N, d_info, 1), "Failed to perform matrix inversion");

    // Check for successful matrix inversion
    checkCudaError(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost), "Failed to copy info to host");
    if (h_info != 0) {
        std::cerr << "Matrix inversion failed" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy the result back to host
    checkCudaError(hipMemcpy(h_Ainv, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy Ainv to host");

    // Print the result
    std::cout << "Inverse matrix:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_Ainv[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_P);
    hipFree(d_info);
    hipblasDestroy(handle);

    return 0;
}