#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define BATCH_COUNT 100

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t err, const char* msg) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << msg << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 1024;
    const int strideA = N * N;
    const int strideB = N * N;
    const int strideC = N * N;

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    float alpha = 1.0f, beta = 0.0f;

    h_A = new float[BATCH_COUNT * N * N];
    h_B = new float[BATCH_COUNT * N * N];
    h_C = new float[BATCH_COUNT * N * N];

    // Initialize matrices
    for (int i = 0; i < BATCH_COUNT * N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
        h_C[i] = 0.0f;
    }

    checkCudaError(hipMalloc((void**)&d_A, BATCH_COUNT * N * N * sizeof(float)), "CUDA malloc A");
    checkCudaError(hipMalloc((void**)&d_B, BATCH_COUNT * N * N * sizeof(float)), "CUDA malloc B");
    checkCudaError(hipMalloc((void**)&d_C, BATCH_COUNT * N * N * sizeof(float)), "CUDA malloc C");

    checkCudaError(hipMemcpy(d_A, h_A, BATCH_COUNT * N * N * sizeof(float), hipMemcpyHostToDevice), "CUDA memcpy A");
    checkCudaError(hipMemcpy(d_B, h_B, BATCH_COUNT * N * N * sizeof(float), hipMemcpyHostToDevice), "CUDA memcpy B");

    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "CUBLAS initialization");

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Event create start");
    checkCudaError(hipEventCreate(&stop), "Event create stop");

    checkCudaError(hipEventRecord(start, 0), "Event record start");

    checkCublasError(
        hipblasSgemmStridedBatched(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            N, N, N,
            &alpha,
            d_A, N, strideA,
            d_B, N, strideB,
            &beta,
            d_C, N, strideC,
            BATCH_COUNT
        ),
        "CUBLAS Strided Batched SGEMM"
    );

    checkCudaError(hipEventRecord(stop, 0), "Event record stop");
    checkCudaError(hipEventSynchronize(stop), "Event synchronize stop");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Event elapsed time");

    std::cout << "cuBLAS Strided Batched GEMM time for " << BATCH_COUNT << " GEMMs: " << milliseconds << " ms" << std::endl;

    // Clean up
    checkCublasError(hipblasDestroy(handle), "CUBLAS destroy");
    checkCudaError(hipFree(d_A), "CUDA free A");
    checkCudaError(hipFree(d_B), "CUDA free B");
    checkCudaError(hipFree(d_C), "CUDA free C");
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
