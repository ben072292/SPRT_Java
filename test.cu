#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA_ERROR(err) (checkCudaError(err, __FILE__, __LINE__))

void checkCudaError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at " << file << ":" << line << " - " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

// Device function for addition
__device__ void add(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

// Device function for multiplication
__device__ void multiply(const float *A, const float *B, float *C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        C[idx] = A[idx] * B[idx];
    }
}

// Kernel to call different device functions based on thread index
__global__ void differentFunctionsKernel(float *A, float *B, float *C_add, float *C_multiply, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    if (idx % 2 == 0) // Even threads perform addition
    {
        add(A, B, C_add, N);
    }
    else // Odd threads perform multiplication
    {
        multiply(A, B, C_multiply, N);
    }
}

int main()
{
    const int N = 16;  // Size of the arrays
    const int SIZE = N * sizeof(float);

    float h_A[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float h_B[N] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    float h_C_add[N];      // Result array for addition
    float h_C_multiply[N]; // Result array for multiplication

    float *d_A, *d_B, *d_C_add, *d_C_multiply;

    // Allocate device memory
    CUDACHECK(hipMalloc(&d_A, SIZE));
    CUDACHECK(hipMalloc(&d_B, SIZE));
    CUDACHECK(hipMalloc(&d_C_add, SIZE));
    CUDACHECK(hipMalloc(&d_C_multiply, SIZE));

    // Copy arrays from host to device
    CUDACHECK(hipMemcpy(d_A, h_A, SIZE, hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(d_B, h_B, SIZE, hipMemcpyHostToDevice));

    // Define block size and grid size
    int blockSize = 16;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Launch the kernel
    differentFunctionsKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C_add, d_C_multiply, N);
    CUDACHECK(hipDeviceSynchronize());

    // Copy result arrays from device to host
    CUDACHECK(hipMemcpy(h_C_add, d_C_add, SIZE, hipMemcpyDeviceToHost));
    CUDACHECK(hipMemcpy(h_C_multiply, d_C_multiply, SIZE, hipMemcpyDeviceToHost));

    // Print the results for addition
    std::cout << "Result of addition:" << std::endl;
    for (int i = 0; i < N; ++i)
    {
        std::cout << h_C_add[i] << " ";
    }
    std::cout << std::endl;

    // Print the results for multiplication
    std::cout << "Result of multiplication:" << std::endl;
    for (int i = 0; i < N; ++i)
    {
        std::cout << h_C_multiply[i] << " ";
    }
    std::cout << std::endl;

    // Clean up
    CUDACHECK(hipFree(d_A));
    CUDACHECK(hipFree(d_B));
    CUDACHECK(hipFree(d_C_add));
    CUDACHECK(hipFree(d_C_multiply));

    return 0;
}